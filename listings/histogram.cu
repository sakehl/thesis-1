#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
typedef DIM1 DimOut;
typedef DIM1 DimIn;
extern "C" __global__ void permute(const DIM1 shIn0, const float* __restrict__ arrIn0_a0, const DIM1 shOut, Int64* __restrict__ arrOut_a0)
{
    const Int64 sh0 = shIn0;
    const DimIn shIn = shape(sh0);
    const int shapeSize = sh0;
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
        DimOut dst;
        const DimIn src = fromIndex(shIn, ix);
        const int v0 = toIndex(shIn0, shape(src));

        dst = (Int64) floorf(arrIn0_a0[v0] / 10.0f);
        if (!ignore(dst)) {
            Int64 y0;
            Int64 _y0;
            const int jx = toIndex(shOut, dst);
            const Int64 v1 = ix;
            const Int64 x0 = (Int64) 1;

            y0 = arrOut_a0[jx];
            do {
                _y0 = y0;
                y0 = atomicCAS64(&arrOut_a0[jx], _y0, x0 + y0);
            } while(y0 != _y0);
        }
    }
}
