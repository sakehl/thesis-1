#include "hip/hip_runtime.h"
/*
 * 1.73:cc: entry function 'generate' used 34 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
 *      ... multiprocessor occupancy 79.7% : 1632 threads over 51 warps in 3 blocksbenchmarking mandelbrot
 *
 * collecting 100 samples, 1 iterations each, in estimated 162.7669 s
 * mean: 16.46181 ms, lb 15.81836 ms, ub 17.53077 ms, ci 0.950
 * std dev: 4.200961 ms, lb 2.716338 ms, ub 6.127911 ms, ci 0.950
 * found 6 outliers among 100 samples (6.0%)
 *   5 (5.0%) high severe
 * variance introduced by outliers: 96.774%
 * variance is severely inflated by outliers
 *
 *
 * After changing the type of booleans to Word32 to avoid a single cvt.u8.u32 in
 * the inner loop:
 *
 * benchmarking mandelbrot
 * collecting 100 samples, 1 iterations each, in estimated 161.6304 s
 * mean: 12.82615 ms, lb 12.51806 ms, ub 13.14113 ms, ci 0.950
 * std dev: 1.593406 ms, lb 1.439425 ms, ub 1.792564 ms, ci 0.950
 * variance introduced by outliers: 85.251%
 * variance is severely inflated by outliers
 */
#include <accelerate_cuda.h>
typedef DIM2 DimOut;
extern "C" __global__ void generate(const DIM0 shIn0, const float* __restrict__ arrIn0_a3, const float* __restrict__ arrIn0_a2, const float* __restrict__ arrIn0_a1, const float* __restrict__ arrIn0_a0, const DIM2 shOut, Word32* __restrict__ arrOut_a0)
{
    const int shapeSize = size(shOut);
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
        const DimOut sh = fromIndex(shOut, ix);
        const Int64 v0 = (Int64) 255;
        const int v1 = toIndex(shIn0, shape());
        const float v2 = arrIn0_a3[v1];
        const float v3 = arrIn0_a2[v1];
        const float v4 = arrIn0_a1[v1];
        const float v5 = arrIn0_a0[v1];
        const Int64 v6 = sh.a1;
        const Int64 v7 = sh.a0;
        const float v8 = v2 + (float) v7 * (v4 - v2) / 800.0f;
        const float v9 = v3 + (float) v6 * (v5 - v3) / 600.0f;
        const int v13 = (Int64) 255;
        float v10 = v8;
        float v11 = v9;
        Int64 v12 = (Int64) 0;

        for (int v19 = 0; v19 < v13; ++v19) {
            const float v14 = v10 * v10 - v11 * v11;
            const float v15 = v10 * v11 + v11 * v10;
            const float v16 = v8 + v14;
            const float v17 = v9 + v15;
            const Word32 v18 = v16 * v16 + v17 * v17 > 4.0f;    // change meeeee

            v10 = v18 ? v10 : v16;
            v11 = v18 ? v11 : v17;
            v12 = v18 ? v12 : (Int64) 1 + v12;
        }

        const Int64 v20 = (Int64) v12;
        const Word32 v21 = v0 == v20;                           // change meeeee
        const Int64 v22 = v0 - v20;
        const Word8 v23 = (Word8) 0;
        const Word8 v24 = (Word8) ((Int64) 7 * v22);
        const Word8 v25 = (Word8) ((Int64) 5 * v22);
        const Word8 v26 = (Word8) ((Int64) 3 * v22);

        arrOut_a0[ix] = v21 ? (Word32) 4278190080 : (Word32) 4294967295 - ((Word32) v23 + (Word32) 256 * (Word32) v24 + (Word32) 65536 * (Word32) v25 + (Word32) 16777216 * (Word32) v26);
    }
}
