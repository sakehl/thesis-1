#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void foldSeg
(
    const DIM1 shIn0,
    const Int32* __restrict__ arrIn0_a0,
    const DIM1 shIn1,
    const float* __restrict__ arrIn1_a0,
    const DIM1 shIn2,
    const Int32* __restrict__ arrIn2_a1,
    const float* __restrict__ arrIn2_a0,
    const DIM1 shOut,
    float* __restrict__ arrOut_a0)
{
    const int vectors_per_block = blockDim.x / warpSize;
    const int num_vectors = vectors_per_block * gridDim.x;
    const int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int vector_id = thread_id / warpSize;
    const int thread_lane = threadIdx.x & warpSize - 1;
    const int vector_lane = threadIdx.x / warpSize;
    const int num_segments = indexHead(shOut);
    const int total_segments = size(shOut);
    int seg;
    int ix;
    extern volatile __shared__ int s_ptrs[][2];
    volatile float* sdata0 = (float*) &s_ptrs[vectors_per_block][2];
    float x0;
    float y0;
    const Int64 sh0 = shIn2;

    for (seg = vector_id; seg < total_segments; seg += num_vectors) {
        const int s = seg % num_segments;
        const int base = seg / num_segments * sh0;

        if (thread_lane < 2) {
            const int v8 = s + thread_lane;

            s_ptrs[vector_lane][thread_lane] = arrIn0_a0[v8];
        }

        const int start = base + s_ptrs[vector_lane][0];
        const int end = base + s_ptrs[vector_lane][1];
        const int num_elements = end - start;

        if (num_elements > warpSize) {
            ix = start - (start & warpSize - 1) + thread_lane;
            if (ix >= start) {
                const Int64 v3 = ix;
                const int v4 = toIndex(shIn2, shape(v3));
                const int v5 = toIndex(shIn1, shape((Int64) arrIn2_a1[v4]));
                const int v6 = toIndex(shIn2, shape(v3));

                y0 = arrIn1_a0[v5] * arrIn2_a0[v6];
            }
            if (ix + warpSize < end) {
                const Int64 v3 = ix + warpSize;
                const int v4 = toIndex(shIn2, shape(v3));
                const int v5 = toIndex(shIn1, shape((Int64) arrIn2_a1[v4]));
                const int v6 = toIndex(shIn2, shape(v3));

                x0 = arrIn1_a0[v5] * arrIn2_a0[v6];
                if (ix >= start) {
                    y0 = x0 + y0;
                } else {
                    y0 = x0;
                }
            }
            for (ix += 2 * warpSize; ix < end; ix += warpSize) {
                const Int64 v3 = ix;
                const int v4 = toIndex(shIn2, shape(v3));
                const int v5 = toIndex(shIn1, shape((Int64) arrIn2_a1[v4]));
                const int v6 = toIndex(shIn2, shape(v3));

                x0 = arrIn1_a0[v5] * arrIn2_a0[v6];
                y0 = x0 + y0;
            }
        } else if (start + thread_lane < end) {
            const Int64 v3 = start + thread_lane;
            const int v4 = toIndex(shIn2, shape(v3));
            const int v5 = toIndex(shIn1, shape((Int64) arrIn2_a1[v4]));
            const int v6 = toIndex(shIn2, shape(v3));

            y0 = arrIn1_a0[v5] * arrIn2_a0[v6];
        }
        ix = min(num_elements, warpSize);
        sdata0[threadIdx.x] = y0;
        if (thread_lane + 32 < ix) {
            x0 = sdata0[threadIdx.x + 32];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 16 < ix) {
            x0 = sdata0[threadIdx.x + 16];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 8 < ix) {
            x0 = sdata0[threadIdx.x + 8];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 4 < ix) {
            x0 = sdata0[threadIdx.x + 4];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 2 < ix) {
            x0 = sdata0[threadIdx.x + 2];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 1 < ix) {
            x0 = sdata0[threadIdx.x + 1];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane == 0) {
            if (num_elements > 0) {
                x0 = 0.0f;
                y0 = x0 + y0;
            } else {
                y0 = 0.0f;
            }
            arrOut_a0[seg] = y0;
        }
    }
}
