#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void foldSeg
(
    const Int64 shIn0_0,
    const float* __restrict__ arrIn0_0,
    const Int64 shIn1_0,
    const Int32* __restrict__ arrIn1_1,
    const float* __restrict__ arrIn1_0,
    const Int64 shIn2_0,
    const Int32* __restrict__ arrIn2_0,
    const Int64 shOut_0,
    float* __restrict__ arrOut_0
)
{
    const int vectors_per_block = blockDim.x / warpSize;
    const int num_vectors = vectors_per_block * gridDim.x;
    const int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int vector_id = thread_id / warpSize;
    const int thread_lane = threadIdx.x & warpSize - 1;
    const int vector_lane = threadIdx.x / warpSize;
    const int num_segments = shOut_0;
    const int total_segments = shOut_0;
    int seg;
    int ix;
    extern volatile __shared__ int s_ptrs[][2];
    volatile float* sdata0 = (float*) &s_ptrs[vectors_per_block][2];
    float x0;
    float y0;
    const Int64 sh0 = shIn1_0;

    for (seg = vector_id; seg < total_segments; seg += num_vectors) {
        const int s = seg % num_segments;
        const int base = seg / num_segments * sh0;

        if (thread_lane < 2) {
            s_ptrs[vector_lane][thread_lane] = arrIn2_0[s + thread_lane];
        }

        const int start = base + s_ptrs[vector_lane][0];
        const int end = base + s_ptrs[vector_lane][1];
        const int num_elements = end - start;

        if (num_elements > warpSize) {
            ix = start - (start & warpSize - 1) + thread_lane;
            if (ix >= start) {
                const Int64 v2 = ({ assert(ix >= 0 && ix < shIn1_0); ix; });
                const Int64 v3 = (Int64) arrIn1_1[v2];

                y0 = arrIn0_0[v3] * arrIn1_0[v2];
            }
            if (ix + warpSize < end) {
                const Int64 v2 = ({ assert(ix + warpSize >= 0 && ix + warpSize < shIn1_0); ix + warpSize; });
                const Int64 v3 = (Int64) arrIn1_1[v2];

                x0 = arrIn0_0[v3] * arrIn1_0[v2];
                if (ix >= start) {
                    y0 = x0 + y0;
                } else {
                    y0 = x0;
                }
            }
            for (ix += 2 * warpSize; ix < end; ix += warpSize) {
                const Int64 v2 = ({ assert(ix >= 0 && ix < shIn1_0); ix; });
                const Int64 v3 = (Int64) arrIn1_1[v2];

                x0 = arrIn0_0[v3] * arrIn1_0[v2];
                y0 = x0 + y0;
            }
        } else if (start + thread_lane < end) {
            const Int64 v2 = ({ assert(start + thread_lane >= 0 && start + thread_lane < shIn1_0); start + thread_lane; });
            const Int64 v3 = (Int64) arrIn1_1[v2];

            y0 = arrIn0_0[v3] * arrIn1_0[v2];
        }
        ix = min(num_elements, warpSize);
        sdata0[threadIdx.x] = y0;
        if (thread_lane + 32 < ix) {
            x0 = sdata0[threadIdx.x + 32];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 16 < ix) {
            x0 = sdata0[threadIdx.x + 16];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 8 < ix) {
            x0 = sdata0[threadIdx.x + 8];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 4 < ix) {
            x0 = sdata0[threadIdx.x + 4];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 2 < ix) {
            x0 = sdata0[threadIdx.x + 2];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane + 1 < ix) {
            x0 = sdata0[threadIdx.x + 1];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (thread_lane == 0) {
            if (num_elements > 0) {
                x0 = 0.0f;
                y0 = x0 + y0;
            } else {
                y0 = 0.0f;
            }
            arrOut_0[seg] = y0;
        }
    }
}

