#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
typedef DIM2 DimOut;
extern "C" __global__ void generate(const DIM2 shIn0, const float* __restrict__ arrIn0_a1, const float* __restrict__ arrIn0_a0, const DIM2 shIn1, const float* __restrict__ arrIn1_a2, const float* __restrict__ arrIn1_a1, const Int64* __restrict__ arrIn1_a0, const DIM2 shOut, float* __restrict__ arrOut_a2, float* __restrict__ arrOut_a1, Int64* __restrict__ arrOut_a0)
{
    const int shapeSize = size(shOut);
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
        const DimOut sh = fromIndex(shOut, ix);
        const int v0 = toIndex(shIn0, shape(sh.a1, sh.a0));
        const float v1 = arrIn0_a1[v0];
        const float v2 = arrIn0_a0[v0];
        const int v3 = toIndex(shIn1, shape(sh.a1, sh.a0));
        const float v4 = arrIn1_a2[v3];
        const float v5 = arrIn1_a1[v3];
        const Int64 v6 = arrIn1_a0[v3];
        const float v7 = v4 * v4 - v5 * v5;
        const float v8 = v4 * v5 + v5 * v4;
        const float v9 = v1 + v7;
        const float v10 = v2 + v8;
        const Word8 v11 = v9 * v9 + v10 * v10 > 4.0f;

        arrOut_a2[ix] = v11 ? v4 : v9;
        arrOut_a1[ix] = v11 ? v5 : v10;
        arrOut_a0[ix] = v11 ? v6 : (Int64) 1 + v6;
    }
}
