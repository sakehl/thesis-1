#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void foldAll(const DIM1 shIn0, const float* __restrict__ arrIn0_a0, const DIM1 shIn1, const float* __restrict__ arrIn1_a0, const DIM0 shOut, float* __restrict__ arrOut_a0)
{
    extern volatile __shared__ float sdata0[];
    float x0;
    float y0;
    const Int64 sh0 = min((Int64) shIn0, (Int64) shIn1);
    const int shapeSize = sh0;
    const int gridSize = blockDim.x * gridDim.x;
    int ix = blockDim.x * blockIdx.x + threadIdx.x;

    if (ix < shapeSize) {
        const Int64 v2 = ix;
        const int v3 = toIndex(shIn0, shape(v2));
        const int v4 = toIndex(shIn1, shape(v2));

        y0 = arrIn0_a0[v3] * arrIn1_a0[v4];
        for (ix += gridSize; ix < shapeSize; ix += gridSize) {
            const Int64 v2 = ix;
            const int v3 = toIndex(shIn0, shape(v2));
            const int v4 = toIndex(shIn1, shape(v2));

            x0 = arrIn0_a0[v3] * arrIn1_a0[v4];
            y0 = x0 + y0;
        }
    }
    sdata0[threadIdx.x] = y0;
    __syncthreads();
    ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
    if (threadIdx.x + 512 < ix) {
        x0 = sdata0[threadIdx.x + 512];
        y0 = y0 + x0;
        sdata0[threadIdx.x] = y0;
    }
    __syncthreads();
    if (threadIdx.x + 256 < ix) {
        x0 = sdata0[threadIdx.x + 256];
        y0 = y0 + x0;
        sdata0[threadIdx.x] = y0;
    }
    __syncthreads();
    if (threadIdx.x + 128 < ix) {
        x0 = sdata0[threadIdx.x + 128];
        y0 = y0 + x0;
        sdata0[threadIdx.x] = y0;
    }
    __syncthreads();
    if (threadIdx.x + 64 < ix) {
        x0 = sdata0[threadIdx.x + 64];
        y0 = y0 + x0;
        sdata0[threadIdx.x] = y0;
    }
    __syncthreads();
    if (threadIdx.x < 32) {
        if (threadIdx.x + 32 < ix) {
            x0 = sdata0[threadIdx.x + 32];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (threadIdx.x + 16 < ix) {
            x0 = sdata0[threadIdx.x + 16];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (threadIdx.x + 8 < ix) {
            x0 = sdata0[threadIdx.x + 8];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (threadIdx.x + 4 < ix) {
            x0 = sdata0[threadIdx.x + 4];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (threadIdx.x + 2 < ix) {
            x0 = sdata0[threadIdx.x + 2];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
        if (threadIdx.x + 1 < ix) {
            x0 = sdata0[threadIdx.x + 1];
            y0 = y0 + x0;
            sdata0[threadIdx.x] = y0;
        }
    }
    if (threadIdx.x == 0) {
        if (shapeSize > 0) {
            if (gridDim.x == 1) {
                x0 = 0.0f;
                y0 = x0 + y0;
            }
            arrOut_a0[blockIdx.x] = y0;
        } else {
            arrOut_a0[blockIdx.x] = 0.0f;
        }
    }
}
