#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64* __restrict__ arrIn0_0, const double* __restrict__ arrIn1_0, Word8* __restrict__ arrOut_0)
{
const int shapeSize = 1;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
arrOut_0[ix] = ({
const Int64 v0 = 0;

;
arrIn1_0[v0];
}) < 1.0e-2 && ({
const Int64 v1 = 0;

;
arrIn0_0[v1];
}) < (Int64) 9999999;
}
}
