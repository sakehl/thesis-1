#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void fold1All(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_3, const double* __restrict__ arrIn0_2, const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, const Int64 shIn1_2, const Int64 shIn1_1, const Int64 shIn1_0, const double* __restrict__ arrIn1_3, const double* __restrict__ arrIn1_2, const double* __restrict__ arrIn1_1, const double* __restrict__ arrIn1_0, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh0 = min(shIn0_2, shIn1_2) * (min(shIn0_1, shIn1_1) * min(shIn0_0, shIn1_0));
const int shapeSize = sh0;
const int gridSize = blockDim.x * gridDim.x;
int ix = blockDim.x * blockIdx.x + threadIdx.x;

/*
         * Reduce multiple elements per thread. The number is determined by the
         * number of active thread blocks (via gridDim). More blocks will result in
         * a larger `gridSize', and hence fewer elements per thread
         *
         * The loop stride of `gridSize' is used to maintain coalescing.
         *
         * Note that we can't simply kill threads that won't participate in the
         * reduction, as exclusive reductions of empty arrays then won't be
         * initialised with their seed element.
         */
if (ix < shapeSize) {
const Int64 v15 = ({ assert(ix >= 0 && ix < min(shIn0_2, shIn1_2) * (min(shIn0_1, shIn1_1) * min(shIn0_0, shIn1_0))); ix; });
const Int64 v16_0 = v15;
const Int64 v16_1 = v16_0 / min(shIn0_0, shIn1_0);
const Int64 v16_2 = v16_1 / min(shIn0_1, shIn1_1);
const Int64 v17 = v16_2 % min(shIn0_2, shIn1_2);
const Int64 v18 = v16_1 % min(shIn0_1, shIn1_1);
const Int64 v19 = v16_0 % min(shIn0_0, shIn1_0);
const Int64 v20 = (v17 * shIn1_1 + v18) * shIn1_0 + v19;
const double v21 = arrIn1_1[v20];
const Word8 v22 = 0.0 == v21;
double lv290;

if (v22) {
lv290 = 1.0e20;
} else {
const Int64 v23 = (v17 * shIn1_1 + v18) * shIn1_0 + v19;
const double v24 = arrIn1_0[v23];

lv290 = v24 / sqrt(({
const Int64 v25 = (v17 * shIn0_1 + v18) * shIn0_0 + v19;
const double v26 = arrIn0_0[v25];

;
v26 * v26;
}) + ({
const Word8 v27 = v21 >= 0.0;
double lv280;

if (v27) {
lv280 = 0.0;
} else {
lv280 = 256.0 * v24 * v24 * v21 * v21;
}
;
lv280;
}));
}
y0 = lv290;
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
const Int64 v15 = ({ assert(ix >= 0 && ix < min(shIn0_2, shIn1_2) * (min(shIn0_1, shIn1_1) * min(shIn0_0, shIn1_0))); ix; });
const Int64 v16_0 = v15;
const Int64 v16_1 = v16_0 / min(shIn0_0, shIn1_0);
const Int64 v16_2 = v16_1 / min(shIn0_1, shIn1_1);
const Int64 v17 = v16_2 % min(shIn0_2, shIn1_2);
const Int64 v18 = v16_1 % min(shIn0_1, shIn1_1);
const Int64 v19 = v16_0 % min(shIn0_0, shIn1_0);
const Int64 v20 = (v17 * shIn1_1 + v18) * shIn1_0 + v19;
const double v21 = arrIn1_1[v20];
const Word8 v22 = 0.0 == v21;
double lv290;

if (v22) {
lv290 = 1.0e20;
} else {
const Int64 v23 = (v17 * shIn1_1 + v18) * shIn1_0 + v19;
const double v24 = arrIn1_0[v23];

lv290 = v24 / sqrt(({
const Int64 v25 = (v17 * shIn0_1 + v18) * shIn0_0 + v19;
const double v26 = arrIn0_0[v25];

;
v26 * v26;
}) + ({
const Word8 v27 = v21 >= 0.0;
double lv280;

if (v27) {
lv280 = 0.0;
} else {
lv280 = 256.0 * v24 * v24 * v21 * v21;
}
;
lv280;
}));
}
x0 = lv290;
z0 = fmin(y0, x0);
y0 = z0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 512 < ix) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
arrOut_0[blockIdx.x] = y0;
}
}
