#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const double* __restrict__ arrIn0_0, const double* __restrict__ arrIn1_0, const double* __restrict__ arrIn2_0, const double* __restrict__ arrIn3_0, double* __restrict__ arrOut_1, double* __restrict__ arrOut_0)
{
const int shapeSize = 1;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 v0 = 0;
const double v1 = arrIn3_0[v0];
const double v17 = fmin(1.0e-2, ({ const Int64 v2 = 0; const double v3 = arrIn0_0[v2]; const Int64 v4 = 0; const double v5 = arrIn2_0[v4]; const Int64 v6 = 0; const double v7 = arrIn1_0[v6]; const Word8 v8 = v7 < 1.0e20; double lv90;  if (v8) { lv90 = 0.5 * v7; } else { lv90 = 1.0e20; }  const Word8 v10 = v5 < lv90; double lv110;  if (v10) { lv110 = 0.6666666666666666 * v5; } else { lv110 = lv90; }  const double v12 = lv110 / v3; const Word8 v13 = v12 >= 1.0 && v12 < 1.1; double lv160;  if (v13) { lv160 = v3; } else { const Word8 v14 = v12 >= 1.0 && v12 > 1.2; double lv150;  if (v14) { lv150 = 1.2 * v3; } else { lv150 = lv110; } lv160 = lv150; } ; lv160; }));
const double v21 = fmin(v17, ({ const double v18 = 1.0e-2 - v1; const Word8 v19 = v18 > v17 && v18 < 4.0 * v17 / 3.0; double lv200;  if (v19) { lv200 = 0.6666666666666666 * v17; } else { lv200 = v18; } ; lv200; }));

arrOut_1[ix] = v1 + v21;
arrOut_0[ix] = v21;
}
}
