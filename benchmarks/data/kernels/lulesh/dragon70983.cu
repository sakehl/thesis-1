#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;

arrOut_0[ix] = 1.0;
}
}
