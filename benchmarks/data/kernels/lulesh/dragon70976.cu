#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void map(const Int64* __restrict__ arrIn0_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = 1;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 x0 = arrIn0_0[ix];

arrOut_0[ix] = (Int64) 1 + x0;
}
}
