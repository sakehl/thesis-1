#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

arrOut_0[ix] = ({
const Int64 v0 = (Int64) 0;
const Int64 v1 = (Int64) 30;

;
({
const Word8 v2 = v0 <= sh2 && (sh2 < v1 && (v0 <= sh1 && (sh1 < v1 && (v0 <= sh0 && sh0 < v1))));
double lv30;

if (v2) {
lv30 = 5.2734375e-5;
} else {
lv30 = 0.0;
}
;
lv30;
}) + ({
const Word8 v5 = v0 <= sh2 && (sh2 < v1 && (v0 <= sh1 && (sh1 < v1 && ({ const Int64 v4 = (Int64) -1 + sh0;  ; v0 <= v4 && v4 < v1; }))));
double lv60;

if (v5) {
lv60 = 5.2734375e-5;
} else {
lv60 = 0.0;
}
;
lv60;
}) + ({
const Word8 v9 = v0 <= sh2 && (sh2 < v1 && ({ const Int64 v7 = (Int64) -1 + sh1;  ; v0 <= v7 && (v7 < v1 && ({ const Int64 v8 = (Int64) -1 + sh0;  ; v0 <= v8 && v8 < v1; })); }));
double lv100;

if (v9) {
lv100 = 5.2734375e-5;
} else {
lv100 = 0.0;
}
;
lv100;
}) + ({
const Word8 v12 = v0 <= sh2 && (sh2 < v1 && ({ const Int64 v11 = (Int64) -1 + sh1;  ; v0 <= v11 && (v11 < v1 && (v0 <= sh0 && sh0 < v1)); }));
double lv130;

if (v12) {
lv130 = 5.2734375e-5;
} else {
lv130 = 0.0;
}
;
lv130;
}) + ({
const Int64 v14 = (Int64) -1 + sh2;
const Word8 v15 = v0 <= v14 && (v14 < v1 && (v0 <= sh1 && (sh1 < v1 && (v0 <= sh0 && sh0 < v1))));
double lv160;

if (v15) {
lv160 = 5.2734375e-5;
} else {
lv160 = 0.0;
}
;
lv160;
}) + ({
const Int64 v17 = (Int64) -1 + sh2;
const Word8 v19 = v0 <= v17 && (v17 < v1 && (v0 <= sh1 && (sh1 < v1 && ({ const Int64 v18 = (Int64) -1 + sh0;  ; v0 <= v18 && v18 < v1; }))));
double lv200;

if (v19) {
lv200 = 5.2734375e-5;
} else {
lv200 = 0.0;
}
;
lv200;
}) + ({
const Int64 v21 = (Int64) -1 + sh2;
const Word8 v24 = v0 <= v21 && (v21 < v1 && ({ const Int64 v22 = (Int64) -1 + sh1;  ; v0 <= v22 && (v22 < v1 && ({ const Int64 v23 = (Int64) -1 + sh0;  ; v0 <= v23 && v23 < v1; })); }));
double lv250;

if (v24) {
lv250 = 5.2734375e-5;
} else {
lv250 = 0.0;
}
;
lv250;
}) + ({
const Int64 v26 = (Int64) -1 + sh2;
const Word8 v28 = v0 <= v26 && (v26 < v1 && ({ const Int64 v27 = (Int64) -1 + sh1;  ; v0 <= v27 && (v27 < v1 && (v0 <= sh0 && sh0 < v1)); }));
double lv290;

if (v28) {
lv290 = 5.2734375e-5;
} else {
lv290 = 0.0;
}
;
lv290;
});
}) / 8.0;
}
}
