#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void map(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_3, const double* __restrict__ arrIn0_2, const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const double x3 = arrIn0_3[ix];
const Word8 v0 = fabs(-1.0 + x3) < 1.0e-7;
double lv10;

if (v0) {
lv10 = 1.0;
} else {
lv10 = x3;
}
arrOut_0[ix] = lv10;
}
}
