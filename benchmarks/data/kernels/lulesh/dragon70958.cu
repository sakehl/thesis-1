#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shIn1_2, const Int64 shIn1_1, const Int64 shIn1_0, const double* __restrict__ arrIn1_23, const double* __restrict__ arrIn1_22, const double* __restrict__ arrIn1_21, const double* __restrict__ arrIn1_20, const double* __restrict__ arrIn1_19, const double* __restrict__ arrIn1_18, const double* __restrict__ arrIn1_17, const double* __restrict__ arrIn1_16, const double* __restrict__ arrIn1_15, const double* __restrict__ arrIn1_14, const double* __restrict__ arrIn1_13, const double* __restrict__ arrIn1_12, const double* __restrict__ arrIn1_11, const double* __restrict__ arrIn1_10, const double* __restrict__ arrIn1_9, const double* __restrict__ arrIn1_8, const double* __restrict__ arrIn1_7, const double* __restrict__ arrIn1_6, const double* __restrict__ arrIn1_5, const double* __restrict__ arrIn1_4, const double* __restrict__ arrIn1_3, const double* __restrict__ arrIn1_2, const double* __restrict__ arrIn1_1, const double* __restrict__ arrIn1_0, const Int64 shIn2_2, const Int64 shIn2_1, const Int64 shIn2_0, const double* __restrict__ arrIn2_2, const double* __restrict__ arrIn2_1, const double* __restrict__ arrIn2_0, const double* __restrict__ arrIn3_0, const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_2, double* __restrict__ arrOut_1, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;
const double v1 = arrIn2_2[v0];
const double v2 = arrIn2_1[v0];
const double v3 = arrIn2_0[v0];
const Int64 v4 = (Int64) 0;
const Int64 v5 = (Int64) -1 + sh1;
const Int64 v6 = (Int64) -1 + sh2;
const Word8 v7 = v4 <= v6 && (v6 < shIn1_0 && (v4 <= v5 && (v5 < shIn1_0 && (v4 <= sh0 && sh0 < shIn1_0))));
double lv90;
double lv91;
double lv92;

if (v7) {
const Int64 v8 = (v6 * shIn1_1 + v5) * shIn1_0 + sh0;

lv92 = arrIn1_2[v8];
lv91 = arrIn1_1[v8];
lv90 = arrIn1_0[v8];
} else {
lv92 = 0.0;
lv91 = 0.0;
lv90 = 0.0;
}

const Int64 v10 = (Int64) -1 + sh0;
const Int64 v11 = (Int64) -1 + sh1;
const Int64 v12 = (Int64) -1 + sh2;
const Word8 v13 = v4 <= v12 && (v12 < shIn1_0 && (v4 <= v11 && (v11 < shIn1_0 && (v4 <= v10 && v10 < shIn1_0))));
double lv150;
double lv151;
double lv152;

if (v13) {
const Int64 v14 = (v12 * shIn1_1 + v11) * shIn1_0 + v10;

lv152 = arrIn1_5[v14];
lv151 = arrIn1_4[v14];
lv150 = arrIn1_3[v14];
} else {
lv152 = 0.0;
lv151 = 0.0;
lv150 = 0.0;
}

const Int64 v16 = (Int64) -1 + sh0;
const Int64 v17 = (Int64) -1 + sh2;
const Word8 v18 = v4 <= v17 && (v17 < shIn1_0 && (v4 <= sh1 && (sh1 < shIn1_0 && (v4 <= v16 && v16 < shIn1_0))));
double lv200;
double lv201;
double lv202;

if (v18) {
const Int64 v19 = (v17 * shIn1_1 + sh1) * shIn1_0 + v16;

lv202 = arrIn1_8[v19];
lv201 = arrIn1_7[v19];
lv200 = arrIn1_6[v19];
} else {
lv202 = 0.0;
lv201 = 0.0;
lv200 = 0.0;
}

const Int64 v21 = (Int64) -1 + sh2;
const Word8 v22 = v4 <= v21 && (v21 < shIn1_0 && (v4 <= sh1 && (sh1 < shIn1_0 && (v4 <= sh0 && sh0 < shIn1_0))));
double lv240;
double lv241;
double lv242;

if (v22) {
const Int64 v23 = (v21 * shIn1_1 + sh1) * shIn1_0 + sh0;

lv242 = arrIn1_11[v23];
lv241 = arrIn1_10[v23];
lv240 = arrIn1_9[v23];
} else {
lv242 = 0.0;
lv241 = 0.0;
lv240 = 0.0;
}

const Int64 v25 = (Int64) -1 + sh1;
const Word8 v26 = v4 <= sh2 && (sh2 < shIn1_0 && (v4 <= v25 && (v25 < shIn1_0 && (v4 <= sh0 && sh0 < shIn1_0))));
double lv280;
double lv281;
double lv282;

if (v26) {
const Int64 v27 = (sh2 * shIn1_1 + v25) * shIn1_0 + sh0;

lv282 = arrIn1_14[v27];
lv281 = arrIn1_13[v27];
lv280 = arrIn1_12[v27];
} else {
lv282 = 0.0;
lv281 = 0.0;
lv280 = 0.0;
}

const Int64 v29 = (Int64) -1 + sh0;
const Int64 v30 = (Int64) -1 + sh1;
const Word8 v31 = v4 <= sh2 && (sh2 < shIn1_0 && (v4 <= v30 && (v30 < shIn1_0 && (v4 <= v29 && v29 < shIn1_0))));
double lv330;
double lv331;
double lv332;

if (v31) {
const Int64 v32 = (sh2 * shIn1_1 + v30) * shIn1_0 + v29;

lv332 = arrIn1_17[v32];
lv331 = arrIn1_16[v32];
lv330 = arrIn1_15[v32];
} else {
lv332 = 0.0;
lv331 = 0.0;
lv330 = 0.0;
}

const Word8 v34 = v4 <= sh2 && (sh2 < shIn1_0 && (v4 <= sh1 && (sh1 < shIn1_0 && (v4 <= sh0 && sh0 < shIn1_0))));
double lv360;
double lv361;
double lv362;

if (v34) {
const Int64 v35 = (sh2 * shIn1_1 + sh1) * shIn1_0 + sh0;

lv362 = arrIn1_23[v35];
lv361 = arrIn1_22[v35];
lv360 = arrIn1_21[v35];
} else {
lv362 = 0.0;
lv361 = 0.0;
lv360 = 0.0;
}

const Int64 v37 = (Int64) -1 + sh0;
const Word8 v38 = v4 <= sh2 && (sh2 < shIn1_0 && (v4 <= sh1 && (sh1 < shIn1_0 && (v4 <= v37 && v37 < shIn1_0))));
double lv400;
double lv401;
double lv402;

if (v38) {
const Int64 v39 = (sh2 * shIn1_1 + sh1) * shIn1_0 + v37;

lv402 = arrIn1_20[v39];
lv401 = arrIn1_19[v39];
lv400 = arrIn1_18[v39];
} else {
lv402 = 0.0;
lv401 = 0.0;
lv400 = 0.0;
}

const double v41 = lv362 + lv402;
const double v42 = lv361 + lv401;
const double v43 = lv360 + lv400;
const double v44 = v41 + lv332;
const double v45 = v42 + lv331;
const double v46 = v43 + lv330;
const double v47 = v44 + lv282;
const double v48 = v45 + lv281;
const double v49 = v46 + lv280;
const double v50 = v47 + lv242;
const double v51 = v48 + lv241;
const double v52 = v49 + lv240;
const double v53 = v50 + lv202;
const double v54 = v51 + lv201;
const double v55 = v52 + lv200;
const double v56 = v53 + lv152;
const double v57 = v54 + lv151;
const double v58 = v55 + lv150;
const double v59 = v56 + lv92;
const double v60 = v57 + lv91;
const double v61 = v58 + lv90;
const Int64 v62 = (sh2 * shIn0_1 + sh1) * shIn0_0 + sh0;
const double v63 = arrIn0_0[v62];
const double v64 = v59 / v63;
const double v65 = v60 / v63;
const double v66 = v61 / v63;
const Word8 v67 = v4 == sh0;
double lv680;

if (v67) {
lv680 = 0.0;
} else {
lv680 = v64;
}

const Word8 v69 = v4 == sh1;
double lv700;

if (v69) {
lv700 = 0.0;
} else {
lv700 = v65;
}

const Word8 v71 = v4 == sh2;
double lv720;

if (v71) {
lv720 = 0.0;
} else {
lv720 = v66;
}

const Int64 v73 = 0;
const double v74 = arrIn3_0[v73];
const double v75 = lv680 * v74;
const double v76 = lv700 * v74;
const double v77 = lv720 * v74;
const double v78 = v1 + v75;
const double v79 = v2 + v76;
const double v80 = v3 + v77;
const Word8 v81 = fabs(v78) < 1.0e-7;
double lv820;

if (v81) {
lv820 = 0.0;
} else {
lv820 = v78;
}

const Word8 v83 = fabs(v79) < 1.0e-7;
double lv840;

if (v83) {
lv840 = 0.0;
} else {
lv840 = v79;
}

const Word8 v85 = fabs(v80) < 1.0e-7;
double lv860;

if (v85) {
lv860 = 0.0;
} else {
lv860 = v80;
}
arrOut_2[ix] = lv820;
arrOut_1[ix] = lv840;
arrOut_0[ix] = lv860;
}
}
