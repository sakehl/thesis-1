#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (Int64) 0;
const Word8 v1 = v0 == sh2 && (v0 == sh1 && v0 == sh0);
double lv20;

if (v1) {
lv20 = 3.948746e7;
} else {
lv20 = 0.0;
}
arrOut_0[ix] = lv20;
}
}
