#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_2, const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, const double* __restrict__ arrIn1_0, const Int64 shIn2_2, const Int64 shIn2_1, const Int64 shIn2_0, const double* __restrict__ arrIn2_2, const double* __restrict__ arrIn2_1, const double* __restrict__ arrIn2_0, const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_2, double* __restrict__ arrOut_1, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;
const double v1 = arrIn2_2[v0];
const double v2 = arrIn2_1[v0];
const double v3 = arrIn2_0[v0];
const Int64 v4 = (sh2 * shIn0_1 + sh1) * shIn0_0 + sh0;
const double v5 = arrIn0_2[v4];
const double v6 = arrIn0_1[v4];
const double v7 = arrIn0_0[v4];
const Int64 v8 = 0;
const double v9 = arrIn1_0[v8];
const double v10 = v5 * v9;
const double v11 = v6 * v9;
const double v12 = v7 * v9;

arrOut_2[ix] = v1 + v10;
arrOut_1[ix] = v2 + v11;
arrOut_0[ix] = v3 + v12;
}
}
