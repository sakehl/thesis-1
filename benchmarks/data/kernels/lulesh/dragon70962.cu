#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shIn1_2, const Int64 shIn1_1, const Int64 shIn1_0, const double* __restrict__ arrIn1_5, const double* __restrict__ arrIn1_4, const double* __restrict__ arrIn1_3, const double* __restrict__ arrIn1_2, const double* __restrict__ arrIn1_1, const double* __restrict__ arrIn1_0, const Int64 shIn2_2, const Int64 shIn2_1, const Int64 shIn2_0, const double* __restrict__ arrIn2_3, const double* __restrict__ arrIn2_2, const double* __restrict__ arrIn2_1, const double* __restrict__ arrIn2_0, const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_1, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Word8 v1 = ({ const Int64 v0 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;  ; arrIn2_1[v0]; }) > 0.0;
double lv580;
double lv581;

if (v1) {
lv581 = 0.0;
lv580 = 0.0;
} else {
const double v5 = ({ const Int64 v2 = (sh2 * shIn0_1 + sh1) * shIn0_0 + sh0;  ; arrIn0_0[v2]; }) / (({ const Int64 v3 = (sh2 * shIn0_1 + sh1) * shIn0_0 + sh0;  ; arrIn0_0[v3]; }) * ({ const Int64 v4 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;  ; arrIn2_3[v4]; }));
const Int64 v6 = (sh2 * shIn1_1 + sh1) * shIn1_0 + sh0;
const double v7 = arrIn1_2[v6];
const double v8 = arrIn1_1[v6];
const double v9 = arrIn1_0[v6];
const Int64 v10 = (sh2 * shIn1_1 + sh1) * shIn1_0 + sh0;
const double v11 = arrIn1_5[v10];
const double v12 = arrIn1_4[v10];
const double v13 = arrIn1_3[v10];
const double v14 = v11 * v7;
const double v15 = v12 * v8;
const double v16 = v13 * v9;
const double v17 = fmin(0.0, v14);
const double v18 = fmin(0.0, v15);
const double v19 = fmin(0.0, v16);
const Int64 v20 = (Int64) 0;
const Int64 v21 = (Int64) 1;
const double v55 = fmin(2.0, fmax(0.0, ({ const double v22 = 1.0 / (1.0e-36 + v7); const double v27 = ({ const Int64 v23 = v21 + sh0; const Word8 v24 = v23 >= shIn0_0 || (sh1 >= shIn0_0 || sh2 >= shIn0_0); double lv260; double lv261; double lv262;  if (v24) { lv262 = 0.0; lv261 = 0.0; lv260 = 0.0; } else { const Int64 v25 = (max(v20, sh2) * shIn1_1 + max(v20, sh1)) * shIn1_0 + max(v20, v23);  lv262 = arrIn1_2[v25]; lv261 = arrIn1_1[v25]; lv260 = arrIn1_0[v25]; } ; lv262; }) * v22;  ; fmin(({ const double v32 = ({ const Int64 v28 = (Int64) -1 + sh0; const Word8 v29 = v28 >= shIn0_0 || (sh1 >= shIn0_0 || sh2 >= shIn0_0); double lv310; double lv311; double lv312;  if (v29) { lv312 = 0.0; lv311 = 0.0; lv310 = 0.0; } else { const Int64 v30 = (max(v20, sh2) * shIn1_1 + max(v20, sh1)) * shIn1_0 + max(v20, v28);  lv312 = arrIn1_2[v30]; lv311 = arrIn1_1[v30]; lv310 = arrIn1_0[v30]; } ; lv312; }) * v22;  ; fmin(v32, 0.5 * (v32 + v27)); }), v27); })));
const double v56 = fmin(2.0, fmax(0.0, ({ const double v33 = 1.0 / (1.0e-36 + v8); const double v38 = ({ const Int64 v34 = v21 + sh1; const Word8 v35 = sh0 >= shIn0_0 || (v34 >= shIn0_0 || sh2 >= shIn0_0); double lv370; double lv371; double lv372;  if (v35) { lv372 = 0.0; lv371 = 0.0; lv370 = 0.0; } else { const Int64 v36 = (max(v20, sh2) * shIn1_1 + max(v20, v34)) * shIn1_0 + max(v20, sh0);  lv372 = arrIn1_2[v36]; lv371 = arrIn1_1[v36]; lv370 = arrIn1_0[v36]; } ; lv371; }) * v33;  ; fmin(({ const double v43 = ({ const Int64 v39 = (Int64) -1 + sh1; const Word8 v40 = sh0 >= shIn0_0 || (v39 >= shIn0_0 || sh2 >= shIn0_0); double lv420; double lv421; double lv422;  if (v40) { lv422 = 0.0; lv421 = 0.0; lv420 = 0.0; } else { const Int64 v41 = (max(v20, sh2) * shIn1_1 + max(v20, v39)) * shIn1_0 + max(v20, sh0);  lv422 = arrIn1_2[v41]; lv421 = arrIn1_1[v41]; lv420 = arrIn1_0[v41]; } ; lv421; }) * v33;  ; fmin(v43, 0.5 * (v43 + v38)); }), v38); })));
const double v57 = fmin(2.0, fmax(0.0, ({ const double v44 = 1.0 / (1.0e-36 + v9); const double v49 = ({ const Int64 v45 = v21 + sh2; const Word8 v46 = sh0 >= shIn0_0 || (sh1 >= shIn0_0 || v45 >= shIn0_0); double lv480; double lv481; double lv482;  if (v46) { lv482 = 0.0; lv481 = 0.0; lv480 = 0.0; } else { const Int64 v47 = (max(v20, v45) * shIn1_1 + max(v20, sh1)) * shIn1_0 + max(v20, sh0);  lv482 = arrIn1_2[v47]; lv481 = arrIn1_1[v47]; lv480 = arrIn1_0[v47]; } ; lv480; }) * v44;  ; fmin(({ const double v54 = ({ const Int64 v50 = (Int64) -1 + sh2; const Word8 v51 = sh0 >= shIn0_0 || (sh1 >= shIn0_0 || v50 >= shIn0_0); double lv530; double lv531; double lv532;  if (v51) { lv532 = 0.0; lv531 = 0.0; lv530 = 0.0; } else { const Int64 v52 = (max(v20, v50) * shIn1_1 + max(v20, sh1)) * shIn1_0 + max(v20, sh0);  lv532 = arrIn1_2[v52]; lv531 = arrIn1_1[v52]; lv530 = arrIn1_0[v52]; } ; lv530; }) * v44;  ; fmin(v54, 0.5 * (v54 + v49)); }), v49); })));

lv581 = -(0.5 * v5 * (v17 * (1.0 - v55) + v18 * (1.0 - v56) + v19 * (1.0 - v57)));
lv580 = 0.6666666666666666 * v5 * (v17 * v17 * (1.0 - v55 * v55) + v18 * v18 * (1.0 - v56 * v56) + v19 * v19 * (1.0 - v57 * v57));
}
arrOut_1[ix] = lv581;
arrOut_0[ix] = lv580;
}
}
