#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void fold1All(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_3, const double* __restrict__ arrIn0_2, const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, const Int64 shOut_0, double* __restrict__ arrOut_0, const Int64 shRec_0, const double* __restrict__ arrRec_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh0 = shRec_0;
const int shapeSize = sh0;
const int gridSize = blockDim.x * gridDim.x;
int ix = blockDim.x * blockIdx.x + threadIdx.x;

/*
         * Reduce multiple elements per thread. The number is determined by the
         * number of active thread blocks (via gridDim). More blocks will result in
         * a larger `gridSize', and hence fewer elements per thread
         *
         * The loop stride of `gridSize' is used to maintain coalescing.
         *
         * Note that we can't simply kill threads that won't participate in the
         * reduction, as exclusive reductions of empty arrays then won't be
         * initialised with their seed element.
         */
if (ix < shapeSize) {
y0 = arrRec_0[ix];
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
x0 = arrRec_0[ix];
z0 = fmin(y0, x0);
y0 = z0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 512 < ix) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
arrOut_0[blockIdx.x] = y0;
}
}
