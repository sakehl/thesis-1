#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shIn1_2, const Int64 shIn1_1, const Int64 shIn1_0, const double* __restrict__ arrIn1_0, const Int64 shIn2_2, const Int64 shIn2_1, const Int64 shIn2_0, const double* __restrict__ arrIn2_1, const double* __restrict__ arrIn2_0, const Int64 shIn3_2, const Int64 shIn3_1, const Int64 shIn3_0, const double* __restrict__ arrIn3_0, const Int64 shIn4_2, const Int64 shIn4_1, const Int64 shIn4_0, const double* __restrict__ arrIn4_3, const double* __restrict__ arrIn4_2, const double* __restrict__ arrIn4_1, const double* __restrict__ arrIn4_0, const Int64 shOut_2, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_3, double* __restrict__ arrOut_2, double* __restrict__ arrOut_1, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_2 * (shOut_1 * shOut_0);
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 tmp_2 = tmp_1 / shOut_1;
const Int64 sh2 = tmp_2 % shOut_2;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const double v1 = fmax(1.0e-9, fmin(1.0e9, ({ const Int64 v0 = (sh2 * shIn4_1 + sh1) * shIn4_0 + sh0;  ; arrIn4_3[v0]; })));
const double v2 = 0.6666666666666666 * (1.0 + (-1.0 + 1.0 / v1));
const Int64 v3 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;
const double v4 = arrIn2_0[v3];
const Int64 v5 = (sh2 * shIn2_1 + sh1) * shIn2_0 + sh0;
const double v6 = arrIn2_1[v5];
const Int64 v7 = (sh2 * shIn0_1 + sh1) * shIn0_0 + sh0;
const double v8 = arrIn0_0[v7];
const Int64 v9 = (sh2 * shIn1_1 + sh1) * shIn1_0 + sh0;
const double v10 = arrIn1_0[v9];
const Int64 v11 = (sh2 * shIn4_1 + sh1) * shIn4_0 + sh0;
const double v12 = arrIn4_2[v11];
const Word8 v13 = v12 > 0.0;
const double v15 = fmax(-1.0e15, ({ const Int64 v14 = (sh2 * shIn3_1 + sh1) * shIn3_0 + sh0;  ; arrIn3_0[v14]; }) - 0.5 * v12 * (v8 + v10));
const double v16 = -1.0 + 1.0 / (v1 - 0.5 * v12);
const double v17 = 0.6666666666666666 * (1.0 + v16);
const double v21 = fmax(0.0, ({ const double v18 = v17 * v15; const Word8 v19 = fabs(v18) < 1.0e-7 || v1 >= 1.0e9; double lv200;  if (v19) { lv200 = 0.0; } else { lv200 = v18; } ; lv200; }));
double lv260;

if (v13) {
lv260 = 0.0;
} else {
lv260 = ({
const double v23 = 0.6666666666666666 * v15 + ({ const double v22 = 1.0 / (1.0 + v16);  ; v22 * v22; }) * v21 * v17;
const Word8 v24 = v23 <= 1.111111e-36;
double lv250;

if (v24) {
lv250 = 3.33333e-19;
} else {
lv250 = sqrt(v23);
}
;
lv250;
}) * v6 + v4;
}

const double v27 = v15 + 0.5 * v12 * (3.0 * (v8 + v10) - 4.0 * (v21 + lv260));
const Word8 v28 = fabs(v27) < 1.0e-7;
double lv290;

if (v28) {
lv290 = 0.0;
} else {
lv290 = fmax(-1.0e15, v27);
}

const double v38 = lv290 - 0.16666666666666666 * v12 * (7.0 * (v8 + v10) - 8.0 * (v21 + lv260) + ({ const double v33 = fmax(0.0, ({ const double v30 = v2 * lv290; const Word8 v31 = fabs(v30) < 1.0e-7 || v1 >= 1.0e9; double lv320;  if (v31) { lv320 = 0.0; } else { lv320 = v30; } ; lv320; }));  ; v33 + ({ double lv370;  if (v13) { lv370 = 0.0; } else { lv370 = ({ const double v34 = 0.6666666666666666 * lv290 + v1 * v1 * v33 * v2; const Word8 v35 = v34 <= 1.111111e-36; double lv360;  if (v35) { lv360 = 3.33333e-19; } else { lv360 = sqrt(v34); } ; lv360; }) * v6 + v4; } ; lv370; }); }));
const Word8 v39 = fabs(v38) < 1.0e-7;
double lv400;

if (v39) {
lv400 = 0.0;
} else {
lv400 = fmax(-1.0e15, v38);
}

const double v44 = fmax(0.0, ({ const double v41 = v2 * lv400; const Word8 v42 = fabs(v41) < 1.0e-7 || v1 >= 1.0e9; double lv430;  if (v42) { lv430 = 0.0; } else { lv430 = v41; } ; lv430; }));
const double v48 = ({ const double v45 = 0.6666666666666666 * lv400 + v1 * v1 * v44 * v2; const Word8 v46 = v45 <= 1.111111e-36; double lv470;  if (v46) { lv470 = 3.33333e-19; } else { lv470 = sqrt(v45); } ; lv470; }) * v6 + v4;
const Word8 v49 = fabs(v48) < 1.0e-7;
double lv500;

if (v49) {
lv500 = 0.0;
} else {
lv500 = v48;
}

const double v51 = 0.6666666666666666 * lv400 + v1 * v1 * v44 * v2;
const Word8 v52 = v51 <= 1.111111e-36;
double lv530;

if (v52) {
lv530 = 3.33333e-19;
} else {
lv530 = sqrt(v51);
}
arrOut_3[ix] = v44;
arrOut_2[ix] = lv400;
arrOut_1[ix] = lv500;
arrOut_0[ix] = lv530;
}
}
