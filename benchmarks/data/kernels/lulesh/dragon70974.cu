#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void map(const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, double* __restrict__ arrOut_0)
{
const int shapeSize = 1;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const double x1 = arrIn0_1[ix];

arrOut_0[ix] = x1;
}
}
