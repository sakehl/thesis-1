#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void fold1All(const Int64 shIn0_2, const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_3, const double* __restrict__ arrIn0_2, const double* __restrict__ arrIn0_1, const double* __restrict__ arrIn0_0, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh0 = shIn0_2 * (shIn0_1 * shIn0_0);
const int shapeSize = sh0;
const int gridSize = blockDim.x * gridDim.x;
int ix = blockDim.x * blockIdx.x + threadIdx.x;

/*
         * Reduce multiple elements per thread. The number is determined by the
         * number of active thread blocks (via gridDim). More blocks will result in
         * a larger `gridSize', and hence fewer elements per thread
         *
         * The loop stride of `gridSize' is used to maintain coalescing.
         *
         * Note that we can't simply kill threads that won't participate in the
         * reduction, as exclusive reductions of empty arrays then won't be
         * initialised with their seed element.
         */
if (ix < shapeSize) {
const Int64 v12 = ({ assert(ix >= 0 && ix < shIn0_2 * (shIn0_1 * shIn0_0)); ix; });
const Int64 v13_0 = v12;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v13_2 = v13_1 / shIn0_1;
const Int64 v14 = v13_2 % shIn0_2;
const Int64 v15 = v13_1 % shIn0_1;
const Int64 v16 = v13_0 % shIn0_0;
const Int64 v17 = (v14 * shIn0_1 + v15) * shIn0_0 + v16;
const double v18 = arrIn0_3[v17];
const double v19 = arrIn0_2[v17];
const double v20 = arrIn0_1[v17];
const double v21 = arrIn0_0[v17];
const Word8 v22 = 0.0 == v20;
double lv230;

if (v22) {
lv230 = 1.0e20;
} else {
lv230 = 0.1 / (1.0e-20 + fabs(v20));
}
y0 = lv230;
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
const Int64 v12 = ({ assert(ix >= 0 && ix < shIn0_2 * (shIn0_1 * shIn0_0)); ix; });
const Int64 v13_0 = v12;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v13_2 = v13_1 / shIn0_1;
const Int64 v14 = v13_2 % shIn0_2;
const Int64 v15 = v13_1 % shIn0_1;
const Int64 v16 = v13_0 % shIn0_0;
const Int64 v17 = (v14 * shIn0_1 + v15) * shIn0_0 + v16;
const double v18 = arrIn0_3[v17];
const double v19 = arrIn0_2[v17];
const double v20 = arrIn0_1[v17];
const double v21 = arrIn0_0[v17];
const Word8 v22 = 0.0 == v20;
double lv230;

if (v22) {
lv230 = 1.0e20;
} else {
lv230 = 0.1 / (1.0e-20 + fabs(v20));
}
x0 = lv230;
z0 = fmin(y0, x0);
y0 = z0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 512 < ix) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmin(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmin(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
arrOut_0[blockIdx.x] = y0;
}
}
